#include "hip/hip_runtime.h"

#include "lut_kernel.h"

#ifdef __LINUX__
	#define __forceinline 
#endif

uchar *dev_a = 0;
uchar *dev_b = 0;
uchar *dev_c = 0;

hipStream_t stream[nStreams];

//int main(){return 0;};

void releaseGPU() {
	releaseCuda();
};

int initGPU(uint size) {
	return (int) initCuda(size);
};

hipError_t releaseCuda(hipError_t cudaStatus) {
	for (int i = 0; i < nStreams; ++i) cudaStatus = hipStreamDestroy(stream[i]);
	hipFree(dev_c);
	hipFree(dev_a);
	//hipFree(dev_b);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return cudaStatus;
	}

	return cudaStatus;
}



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

#ifdef USE_DEVICE_CODE
__device__ __forceinline void device_LUT(uchar*c, uint size) {
	unsigned int i = __fmaf_rd(__fmaf_rd(blockIdx.x,blockDim.x,threadIdx.x),3,0);
	if (i >= size) return; // out of bounds
	// get color value
	unsigned int r = (i >> 16) & 0xFF;
	unsigned int g = (i >> 8) & 0xFF;
	unsigned int b = i & 0xFF;

	// make calculations
	c[i] = __fmaf_rd(r, 1, 100);
	c[i+1] = __fmaf_rd(g, 1, 100);
	c[i+2] = __fmaf_rd(b, 1, 100);
}
#endif

__global__ void applyLUTGPUKernel(uchar *dest, uchar *ori, uint size, uchar *LUT) {
	unsigned int i = (blockIdx.x*blockDim.x + threadIdx.x);
	if (i >= size) return; // out of bounds
	i *= 3;
	uint idx = (ori[i+2] << 16) | (ori[i + 1] << 8) | ori[i];
	dest[i] = (uchar)LUT[idx];
	dest[i+1] = (uchar)LUT[idx+1];
	dest[i+2] = (uchar)LUT[idx+2];
	/*

	dest[i] = min((uchar)ori[i]+100, 255);
	dest[i + 1] = min((uchar)ori[i+1], 255);
	dest[i + 2] = min((uchar)ori[i+2]+100, 255);
	*/
}

__global__ void makeLUTGPUKernel(uchar *c, uint size)
{

//RobôCIn
	/*unsigned int i = (blockIdx.x*blockDim.x + threadIdx.x);
	if (i >= size) return; // out of bounds
	// get color value
	uchar r = (i >> 16) & 0xFF;
	uchar g = (i >> 8) & 0xFF;
	uchar b = i & 0xFF;

	// make calculations
	color.y = (9798 * r + 19235 * g + 3736 * b) >> 15;
    color.u = (18514 * (b - color.y) >> 15) + 128;
    color.v = (23364 * (r - color.y) >> 15) + 128;

    index = r * 65536 + g * 256 + b;

	@todo: substitute the this->getColorLabel(color) call
	c[i] = this->getColorLabel(color);*/

//old
#ifdef USE_DEVICE_CODE
	//device_LUT(c, size);
	unsigned int i = __fmaf_rd(__fmaf_rd(blockIdx.x, blockDim.x, threadIdx.x), 3, 0);
	if (i >= size) return; // out of bounds
						   // get color value
	unsigned int r = (i >> 16) & 0xFF;
	unsigned int g = (i >> 8) & 0xFF;
	unsigned int b = i & 0xFF;

	// make calculations
	c[i] = __fmaf_rd(r, 1, 100);
	c[i + 1] = __fmaf_rd(g, 1, 100);
	c[i + 2] = __fmaf_rd(b, 1, 100);
#else
	unsigned int i = (blockIdx.x*blockDim.x + threadIdx.x);
	if (i >= size) return; // out of bounds
	// get color value
	uchar r = (i >> 16) & 0xFF;
	uchar g = (i >> 8) & 0xFF;
	uchar b = i & 0xFF;
	i *= 3;
	// make calculations
	c[i] = min((uchar)b,255);
	c[i+1] = min((uchar)g,255);
	c[i+2] = min((uchar)r+100,255);
#endif
}

int makeLUTGPU(uint lut_size)
{

    // create LUT with CUDA
    hipError_t cudaStatus = makeLUTWithCuda(lut_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "makeLUTWithCuda failed!");
        return 1;
    }


    return 0;
}

hipError_t initCuda(unsigned int size) {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(uchar) * 3);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(uchar) * 3);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));


	for (int i = 0; i < nStreams; ++i) {
		cudaStatus = hipStreamCreate(&stream[i]);
		//hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
	}
		
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	return cudaStatus;
Error:
	return releaseCuda(cudaStatus);
}

// Helper function for using CUDA to make LUT in parallel.
hipError_t makeLUTWithCuda(uint size)
{

	hipError_t cudaStatus;

	uint blockSize = CUDA_BLOCK_SIZE;
	uint gridSize = (size + blockSize-1) / blockSize;
	//gridSize = min(gridSize, 100);
	//fprintf(stderr,"grid, block %u %u\n", gridSize, blockSize);
	// Launch a kernel on the GPU with one thread for each element.
	makeLUTGPUKernel <<< gridSize, blockSize >>>(dev_c, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "makeLUTGPUKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	return cudaStatus;
Error:
	return releaseCuda(cudaStatus);
}



// Helper function for using CUDA to make LUT in parallel.
hipError_t applyLUTWithCuda(uchar *c, uint size)
{

	hipError_t cudaStatus;

	hipMemcpy(dev_a, c, size*3, hipMemcpyHostToDevice);
	/*cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "memcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}*/
	uint blockSize = CUDA_BLOCK_SIZE;
	uint gridSize = (size + blockSize - 1) / blockSize;
	//gridSize = min(gridSize, 100);
	//fprintf(stderr, "grid, block %u %u\n", gridSize, blockSize);
	// Launch a kernel on the GPU with one thread for each element.
	applyLUTGPUKernel << < gridSize, blockSize >> >(dev_a, dev_a, size, dev_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "applyLUTGPUKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	/*cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}*/
	hipMemcpy(c, dev_a, size * 3, hipMemcpyDeviceToHost);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "memcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	return cudaStatus;
Error:
	return releaseCuda(cudaStatus);
}

void applyLUTGPU(uchar *c, uint size) {
	applyLUTWithCuda(c, size);
}

hipError_t getLUTFromGPUCUDA(uchar *c, uint size) {
	hipError_t cudaStatus;
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(uchar), hipMemcpyDeviceToHost);
	//cudaStatus = hipMemcpyAsync(c, dev_c, size * sizeof(uchar), hipMemcpyDeviceToHost);
	/*int streamSize = (size) / nStreams;
	for (int i = 0; i < nStreams; ++i) {
		uint offset = i * streamSize;
		uint streamBytes = max(size - offset, 0);
		hipMemcpyAsync(c+offset, dev_c+offset, streamBytes, hipMemcpyDeviceToHost, stream[i]);
	}

	for (int i = 0; i < nStreams; ++i) {
		hipStreamSynchronize(stream[i]);
	}*/

	cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	return cudaStatus;
}

int getLUTFromGPU(uchar *c, uint size) {
	if (getLUTFromGPUCUDA(c, size) != hipSuccess) return 1;
	return 0;
}

hipError_t setLUTToGPUCUDA(uchar *c, uint size) {
	hipError_t cudaStatus;
	// Copy output vector from host memory to GPU buffer.
	cudaStatus = hipMemcpy(dev_c,c , size * sizeof(uchar), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpyAsync(dev_c, c, size * sizeof(uchar), hipMemcpyHostToDevice);
	/*int streamSize = (size) / nStreams;
	for (int i = 0; i < nStreams; ++i) {
		uint offset = i * streamSize;
		uint streamBytes = max(size - offset, 0);
		hipMemcpyAsync(dev_c+offset, c+offset, streamBytes, hipMemcpyHostToDevice, stream[i]);
	}

	for (int i = 0; i < nStreams; ++i) {
		hipStreamSynchronize(stream[i]);
	}*/

	cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	return cudaStatus;
}

int setLUTToGPU(uchar *c, uint size) {
	if (setLUTToGPUCUDA(c, size) != hipSuccess) return 1;
	return 0;
}
//
//// Helper function for using CUDA to make LUT in parallel.
//hipError_t LUTWithCuda(uchar *c, uchar *a, unsigned int size)
//{
//  
//	hipError_t cudaStatus;
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(uchar)*3, hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(uchar)*3, hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//    return cudaStatus;
//Error:
//	return releaseCuda(cudaStatus);
//}
